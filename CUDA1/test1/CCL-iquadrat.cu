#include "hip/hip_runtime.h"
#include <string.h>
#include <iostream>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <vector>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <climits>
#include <QImage>
#ifndef TIMER_H
#define TIMER_H


class Timer {
  
public:
    
    static double now() {
        timeval now;
        gettimeofday(&now, 0);
        return now.tv_sec + now.tv_usec/1000000.0;
    }
    
    static long nowMilisecs() {
        timeval now;
        gettimeofday(&now, 0);
        return (long) (now.tv_sec*1000 + now.tv_usec/1000.0);
    }
    
    Timer() {
        elapsed = 0;
        running = false;
    }
    
    void start() {
        running = true;
        gettimeofday(&started, 0);
    }
    
    void restart() {
      running = true;
      elapsed = 0;
      gettimeofday(&started, 0);
    }
    
    double getTimeElapsed() {
           return elapsed + ( (running) ? timeSinceStart() : 0 ); 
    }
    
    double stopAndAddTime() {
       double dt = timeSinceStart();
       if (dt <0) {
           //exit(9);
       }
       elapsed += dt;
       running = false;
       return elapsed;
    }
    
    void reset() {
        elapsed = 0;
        running = false;
    }

private:
    
    double timeSinceStart() {
        timeval now;
        gettimeofday(&now, 0);
        return (now.tv_sec - started.tv_sec) + (now.tv_usec - started.tv_usec)/1000000.0;
    }

    timeval started;    
    volatile double  elapsed;
    volatile bool    running;    
    
};

#endif

#define PRINT_TIMINGS
#define FIND_VARIANT1

using std::vector;

enum {
    NW = 1, N = 4, NE = 5,
    W  = 0
};

inline void resize_non_initialized(vector<int>& v, int size)
{
    v.reserve(size);
    ((int**)&v)[1] = ((int**)&v)[0]+size;
}


class CCL {


    unsigned int v[4] __attribute__ ((aligned(16)));    
    
    vector<int>* m_result;
    vector<int>* m_img;
    int m_W;
    int m_H;
    int m_conn;
    int m_thresh;
    int m_thresh_p1;    
    bool m_diag;
    Timer t;
    
public:
     inline vector<int> cuda_ccl(vector<int>& img, int W, int conn, int threshold) {
     
        vector<int> result_vec;    
                
#ifdef PRINT_TIMINGS        
        printf("start\n");
#endif        
//        printf("gcc version %d.%d.%d\n",__GNUC__,__GNUC_MINOR__,__GNUC_PATCHLEVEL__);
        
        t.start();
        m_W = W;
        m_H = img.size() / W;
        m_conn = conn;
        m_thresh = threshold;
        m_thresh_p1 = threshold+1;
        m_img = &img;
        m_diag = (m_conn == 8);                

        m_result = &img;    
//        m_result = new vector<int>(m_W*m_H);
//    m_result = &result_vec;
//        resize_non_initialized(result_vec, m_W*m_H);
        
        internal_cuda_ccl();

#ifdef PRINT_TIMINGS
        printf("t total = %f\n", t.getTimeElapsed());
        fprintf(stderr, "done in %f, (stderr)\n", t.getTimeElapsed());
        
        fflush(stdout);
        fflush(stdin);
#endif        

        result_vec.swap(*m_result);
        return result_vec;
    }
    
    void internal_cuda_ccl();

private:


    
    void cuda_neighbour_process();    

    void forward_pass();
    
    void final_pass();
    
    inline void compute_min(int src, int nv1);
    inline void compute_min(int src, int nv1, int nv2);    
    inline void compute_min(int src, int nv1, int nv2, int nv3);    
    inline void compute_min(int src, int nv1, int nv2, int nv3, int nv4);        
    
    inline int get_neigh(int idx, int off, int dir);
    
    inline int img(int x, int y);
    
  inline int& src_img(int x, int y) {
        return m_img->operator[](y*m_W + x);
    }
    
    inline int& result_img(int x, int y) {
        return m_result->operator[](y*m_W + x);
    }
    
    inline int& result_img(int off) {
        return m_result->operator[](off);
    }
    
    inline bool is_conn(int x1, int y1, int dir);
    
    inline int is_conn(int* p1, int* p2) {

#ifdef HAVE_INTRINSICS    
      __m128i i1 = _mm_set_epi32(0,0,0,*p1);
        __m128i i2 = _mm_set_epi32(0,0,0,*p2);
        
        __m128i d  = _mm_sad_epu8(i1,i2);
        __m128i t  = _mm_set_epi32(0,0,0,m_thresh_p1);
        __m128i c  = _mm_cmplt_epi32(d,t);        
        return _mm_movemask_epi8(c) & 1;
#else 

    // PRECONDITION m_thresh_p1 in %xmm7

    int result;
     
    __asm__ __volatile__(
        "movd    (%[v1]), %%xmm1 \n\t"
            "movd    (%[v2]), %%xmm0 \n\t"
            "psadbw    %%xmm0, %%xmm1 \n\t"
      "movdqa %%xmm7, %%xmm0 \n\t"
            "pcmpgtd    %%xmm1, %%xmm0 \n\t"            
            "pmovmskb    %%xmm0, %0 \n\t"
            "andl    $1, %0"
          : "=r"(result)
            : [v1] "r" (p1), [v2] "r" (p2), [t] "r" (&m_thresh_p1)
            : 
        );
        
        return result;

#endif        
    }
    
    
    inline int dist(int v1, int v2);
    
    inline void unite(int g1, int g2);
    
    inline int find0(int v);
    
    inline int find1(int v);
    
    inline void findAndUnite(int v1, int v2);    
    
    inline void findAndUniteIfConnected(int v1, int v2);    
    
    int blue(int v);
    
    int green(int v);    
    
    int red(int v);        
    
};



#ifdef FIND_VARIANT1
int CCL::find1(int x) {
    int c = x;
    int p = (*m_result)[c];
    int gp = (*m_result)[p];
    if (p == gp) return p;
    
    while(gp != p) {
        (*m_result)[p] = gp;
        c = p;
        p = gp;
        gp = (*m_result)[gp];
    }
    return p;
}
#endif

#ifdef FIND_VARIANT2
int CCL::find1(int v) {
  int c  = v;
    int p  = (*m_result)[c];
    int gp = (*m_result)[p];
    if (p == gp) return p;
    
  do {
      (*m_result)[c] = gp;
        c = p;
        p = gp;
        gp = (*m_result)[gp];
    } while (p != gp);
  (*m_result)[c] = gp;    
  return p;
}
#endif

int CCL::find0(int v) {
  int c = (*m_result)[v];
    int p = (*m_result)[c];
    if (c == p) return c;
  do {
        c = p;
        p = (*m_result)[c];
    } while(c != p);
    return c;
}


void CCL::findAndUnite(int v1, int v2) {
    int g1 = find0(v1);
    int g2 = find0(v2);
    
    if (g1 < g2) {
      unite(g1, g2);
    } else if (g2 < g1) {
      unite(g2, g1);
    }
}

void CCL::findAndUniteIfConnected(int v1, int v2) {
  if (!is_conn(&(*m_img)[v1], &(*m_img)[v2])) return;
    findAndUnite(v1, v2);
}
    
// g1 <= g2!
void CCL::unite(int g1, int g2) {
    (*m_result)[g2] = g1;
} 

//#define HAVE_INTRINSICS


#ifdef HAVE_INTRINSICS

void debug(__m128i v) {
    unsigned int f[4];
    _mm_store_si128((__m128i*)&f[0], v);
    printf("<%u, %u, %u, %u>", f[0], f[1], f[2], f[3]);
  }

#endif

using std::vector;
using std::cout;
using std::endl;


inline unsigned int min2(unsigned int a, unsigned int b) {
    return a < b ? a : b;
}

unsigned int getCpuId() {

  unsigned int a,b,c,d;    
    unsigned int v = 1;
    __asm__ __volatile__ ("cpuid" : "=a" (a), "=b" (b), "=c" (c), "=d" (d) : "a" (v));
    
    std::cout << a << "/" << b << "/" << c << "/" << d << std::endl;
    return c;
}

#ifdef HAVE_SSE4
    #define PMIN10 "pminud  %%xmm1, %%xmm0 \n\t"
#else
    #define PMIN10  \
        "movdqa   %%xmm1, %%xmm4 \n\t" \
        "movdqa   %%xmm1, %%xmm5 \n\t" \
        "pcmpgtd  %%xmm0, %%xmm5 \n\t" \
        "pand     %%xmm5, %%xmm0 \n\t" \
        "pandn    %%xmm4, %%xmm5 \n\t" \
        "por      %%xmm5, %%xmm0 \n\t" 
#endif

#ifdef HAVE_SSE4
    #define PMIN03 "pminud  %%xmm0, %%xmm3 \n\t"
#else
    #define PMIN03  \
        "movdqa   %%xmm0, %%xmm4 \n\t" \
        "movdqa   %%xmm0, %%xmm5 \n\t" \
        "pcmpgtd  %%xmm3, %%xmm5 \n\t" \
        "pand     %%xmm5, %%xmm3 \n\t" \
        "pandn    %%xmm4, %%xmm5 \n\t" \
        "por      %%xmm5, %%xmm3 \n\t" 
#endif

inline void min4(unsigned int* v) {

//    std::cout << "min4 " << v[0] << "," << v[1] << "," << v[2] << "," << v[3] << std::endl;
#ifdef ASM_MIN4
  __asm__ __volatile__ (
      
        "movdqa (%[pv]), %%xmm0 \n\t"
        "pshufd $14, %%xmm0, %%xmm1\n\t"
        PMIN10
        "pshufd $1, %%xmm0, %%xmm1\n\t"
        PMIN10
        "movdqa %%xmm0, (%[pv]) \n\t"
        :
        : [pv] "r" (v)
        : "memory"
    );
    
//    std::cout << "->  " << v[0] << "," << v[1] << "," << v[2] << "," << v[3] << std::endl;
    
//    std::cout << "min=" << v[0] << endl;
#else    
    int ab = min2(v[0],v[1]);
    int cd = min2(v[2],v[3]);
    v[0] = min2(ab, cd);
#endif    
}

void test_min4(int a, int b, int c, int d) {

    unsigned int v[4] __attribute__ ((aligned(16)));    
    v[0] = a;
    v[1] = b;
    v[2] = c;
    v[3] = d;    
    min4(v);        
    
    cout << "->" << v[0] << endl;
    
}

inline void min(int count, unsigned int* v) {
    if (count == 1) return;
    if (count == 2) {
        v[0] = min2(v[0], v[1]);
    } else if (count == 3) {
        v[0] = min2(min2(v[0], v[1]), v[2]);
    } else if (count == 4) {
        min4(v);
    } else {
        exit(6);
    }
}



void CCL::compute_min(int src, int nv1) {
  int g1 = find1(nv1);
    unite(g1, src);
}

void CCL::compute_min(int src, int nv1, int nv2) {
  int g1 = find1(nv1);
    int g2 = find1(nv2);
    int gmin = min2(g1, g2);
    unite(gmin, g1);
    unite(gmin, g2);
    unite(gmin, src);
}

void CCL::compute_min(int src, int nv1, int nv2, int nv3) {
  int g1 = find1(nv1);
    int g2 = find1(nv2);
    int g3 = find1(nv3);    
    int gmin = min2(min2(g1, g2), g3);
    unite(gmin, g1);
    unite(gmin, g2);
    unite(gmin, g3);    
    unite(gmin, src);
}

void CCL::compute_min(int src_v, int nv1, int nv2, int nv3, int nv4) {
  int g1 = find1(nv1);
    int g2 = find1(nv2);
    int g3 = find1(nv3);    
    int g4 = find1(nv4);    
    int gmin = min2(min2(g1, g2), min2(g3,g4));
    unite(gmin, g1);
    unite(gmin, g2);
  unite(gmin, g3);
  unite(gmin, g4);    
    unite(gmin, src_v);
}

void CCL::internal_cuda_ccl() {
    
      cout << "width=" << m_W << " height=" << m_H << endl;
        cout << "thresh=" << m_thresh << " conn=" << m_conn << endl;
        
        cout << "cpuid=" << getCpuId() << endl;
    
#ifndef HAVE_INTRINSICS
      cout << "using asm version" << endl;
   __asm__ __volatile__ (
        "movd    (%[t]), %%xmm7 \n\t"
            "pshufd $68, %%xmm7,  %%xmm7 \n\t"
            :
            : [t] "r" (&m_thresh_p1)
            :
     );
#endif         

        cuda_neighbour_process();
                
      Timer t_step;
        
/*        t_step.start();
      forward_pass();
        cout << " time for union     : " <<  t_step.getTimeElapsed() << endl;
*/        
/*        t_step.restart();
        final_pass();
        cout << " time for final find: " <<  t_step.getTimeElapsed() << endl;
*/        
}


void CCL::forward_pass() {
      int size = m_H * m_W;      
        
        int img_nw = -m_W-1;
        int img_w = -1;        

        for(int i = 0; i<size; ++i) {
            int magic = (*m_img)[img_w+1];
                        
                switch(magic) {
                  case 0: {
                    (*m_img)[img_w+1] = img_w+1;
                        break;
                    }                    
                    case 2: {
                        compute_min(img_w+1, img_nw);
                    } break;                    
                    case 16: {
                        compute_min(img_w+1, img_nw+1);
                    } break;
                    case 18: {
                        compute_min(img_w+1, img_nw, img_nw+1);
                    } break;
                    case 32: {
                        compute_min(img_w+1, img_nw+2);
                    } break;
                    case 34: {
                        compute_min(img_w+1, img_nw, img_nw+2);                    
                    } break;
                    case 48: {
                        compute_min(img_w+1, img_nw+1, img_nw+2);
                    } break;
                    case 50: {
                        compute_min(img_w+1, img_nw, img_nw+1, img_nw+2);
                    } break;
                    case 1: {
                        compute_min(img_w+1, img_w);
                    } break;
                    case 3: {
                        compute_min(img_w+1, img_nw, img_w);
                    } break;
                    case 17: {
                        compute_min(img_w+1, img_nw+1, img_w);
                    } break;
                    case 19: {
                        compute_min(img_w+1, img_nw, img_nw+1, img_w);
                    } break;
                    case 33: {
                        compute_min(img_w+1, img_nw+2, img_w);
                    } break;
                    case 35: {
                        compute_min(img_w+1, img_nw, img_nw+2, img_w);                    
                    } break;
                    case 49: {
                        compute_min(img_w+1, img_nw+1, img_nw+2, img_w);                    
                    } break;
                    case 51: {
                        compute_min(img_w+1, img_nw, img_nw+1, img_nw+2, img_w);
                    } break;                
/*                     default:
                    {
                        printf("Oops, got magic number %d with i=%d\n", magic, i);
                    }
 */                     
            }
            
            img_nw++;
            img_w++;
        }
}
    
void CCL::final_pass() {
    int size = m_W * m_H;
    for(int i=0; i<size; ++i) {
        (*m_result)[i] = find0(i);    
    }
}    
        
    int CCL::img(int x, int y) {
        return (*m_img)[y*m_W + x];
    }

    int CCL::dist(int v1, int v2) {
        return abs(red(v1) - red(v2)) + abs(green(v1)- green(v2)) + abs(blue(v1)-blue(v2));    
    }
    
    int CCL::blue(int v) {
        return v & 0xFF; 
    }
    
    int CCL::green(int v) {
        return (v >> 8) & 0xFF;
    }
    
    
    int CCL::red(int v) {
        return (v >> 16) & 0xFF;
    }            

//#define USE_PATH_SHORTENING

// local ccl thread count must be a multiple of BLOCK_SIZE_X and a divider of BLOCK_SIZE_X*BLOCK_SIZE_Y

#define BLOCK_SIZE_X (44)
#define BLOCK_SIZE_Y (44)

#define LOCAL_CCL_THREADS (484)
#define LOCAL_CCL_Y_STEP (LOCAL_CCL_THREADS / BLOCK_SIZE_X)
#define LOCAL_CCL_Y_IT (BLOCK_SIZE_X * BLOCK_SIZE_Y / LOCAL_CCL_THREADS)

#define FINAL_BLOCK_SIZE_X (32)
#define FINAL_BLOCK_SIZE_Y (32)

#define FINAL_PASS_THREADS (512)
#define FINAL_PASS_Y_STEP (FINAL_PASS_THREADS / FINAL_BLOCK_SIZE_X)
#define FINAL_PASS_Y_IT (FINAL_BLOCK_SIZE_X * FINAL_BLOCK_SIZE_Y / FINAL_PASS_THREADS)

__device__ bool is_conn(int v1, int v2, int t) {
/*    int sum = __usad((v1 & 0x0000FF)>>0, (v2 & 0x0000FF)>>0,  0);
        sum = __usad((v1 & 0x00FF00)>>8, (v2 & 0x00FF00)>>8,  sum);
        sum = __usad((v1 & 0xFF0000)>>16,(v2 & 0xFF0000)>>16, sum);    */
            
    int sum  = __usad((v1 & 0xFF0000), (v2 & 0xFF0000), 0) >> 16;
        sum += __usad((v1 & 0x00FF00), (v2 & 0x00FF00), 0) >> 8;                    
        sum  = __usad((v1 & 0x0000FF), (v2 & 0x0000FF), sum);

    return sum <= t;    
}

__device__ int find(int* buf, int x) {
#ifdef USE_PATH_SHORTENING
    int c = x;
    int p = buf[c];
    int gp = buf[p];
    if (p == gp) return p;
    
    while(gp != p) {
        buf[p] = gp;
        c = p;
        p = gp;
        gp = buf[gp];
    }
    return p;
#else    
    
    while (x != buf[x]) {
      x = buf[x];
    }
    return x;
    
#endif
}


__device__ void findAndUnion(int* buf, int g1, int g2) {
    bool done;    
    do {

         g1 = find(buf, g1);
      g2 = find(buf, g2);    
            
      // it should hold that g1 == buf[g1] and g2 == buf[g2] now
    
        if (g1 < g2) {
          int old = atomicMin(&buf[g2], g1);
            done = (old == g2);
            g2 = old;
        } else if (g2 < g1) {
          int old = atomicMin(&buf[g1], g2);
            done = (old == g1);
            g1 = old;
        } else {
          done = true;
        }
        
    } while(!done);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(74);
    }                         
}

__global__ void join_seams4_cuda(int* d_img, int* d_dst_img, int w, int t, int h) {

    int offset = BLOCK_SIZE_Y * blockIdx.y * w;

    // join x seams

    int ymax = h - blockIdx.y * BLOCK_SIZE_Y;
    if (ymax > BLOCK_SIZE_Y) ymax = BLOCK_SIZE_Y;    

  int n = threadIdx.x * gridDim.x + blockIdx.x;
//  int n = blockIdx.x * blockDim.x + threadIdx.x;

    int x = n / BLOCK_SIZE_Y;
    int y = n % BLOCK_SIZE_Y;

    int idx = offset + y * w + x * BLOCK_SIZE_X;

    if (x > 0 && y < ymax && is_conn(d_img[idx], d_img[idx - 1], t)) {
      findAndUnion(d_dst_img, idx, idx - 1);
    }

    // join y seams

    if (offset != 0) {

        int x = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;
        int idx = offset + x;

        if (x < w && is_conn(d_img[idx], d_img[idx-w], t)) {
          findAndUnion(d_dst_img, idx, idx - w);
        }    

    }
        
}

__global__ void join_seams8_cuda(int* d_img, int* d_dst_img, int w, int t, int h) {
  
    int offset = BLOCK_SIZE_Y * blockIdx.y * w;        
    
    int ymax = h - blockIdx.y * BLOCK_SIZE_Y;
    if (ymax > BLOCK_SIZE_Y) ymax = BLOCK_SIZE_Y;        

    // join x seams    

  int n = threadIdx.x * gridDim.x + blockIdx.x;
//  int n = blockIdx.x * blockDim.x + threadIdx.x;

    int x = n / BLOCK_SIZE_Y;
    int y = n % BLOCK_SIZE_Y;

    int idx = offset + y * w + x * BLOCK_SIZE_X;

    if (x > 0 && y < ymax) {

        if (is_conn(d_img[idx], d_img[idx - 1], t)) {
          findAndUnion(d_dst_img, idx, idx - 1);
        }

        if (y > 0 && is_conn(d_img[idx], d_img[idx - 1 - w], t)) {
          findAndUnion(d_dst_img, idx, idx - 1 - w);
        }

        if (y > 0 && is_conn(d_img[idx - 1], d_img[idx - w], t)) {
          findAndUnion(d_dst_img, idx - 1, idx - w);
        }

    }

    // join y seams

    x = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;

    if (offset != 0 && x < w) {

        int idx = offset + x;

        if (is_conn(d_img[idx], d_img[idx - w], t)) {
          findAndUnion(d_dst_img, idx, idx - w);
        }

        if (x > 0 && is_conn(d_img[idx], d_img[idx - w - 1], t)) {
          findAndUnion(d_dst_img, idx, idx - w - 1);
        }
        if (x > 0 && is_conn(d_img[idx - 1], d_img[idx - w], t)) {
          findAndUnion(d_dst_img, idx - 1, idx - w);
        }

    }    

}


__global__ void final_find_cuda(int* d_dst_img, int w, int h) {
    
    int x = FINAL_BLOCK_SIZE_X * blockIdx.x + threadIdx.x;
    
    for(int i=0; i<FINAL_PASS_Y_IT; ++i) {
      int y = FINAL_BLOCK_SIZE_Y * blockIdx.y + FINAL_PASS_Y_STEP * i + threadIdx.y;
        if (x < w && y < h) {
          int idx = y * w + x;
            d_dst_img[idx] = find(d_dst_img, idx);
        }
    }

}

__global__ void calculate_neighbours4_cuda(int* d_img, int* d_dst_img, int w, int t, int h) {

  extern __shared__ int s_buf[];
    
    int* s_buf2 = &s_buf[BLOCK_SIZE_X * BLOCK_SIZE_Y];
    
    int offset = BLOCK_SIZE_Y * blockIdx.y * w;
  int global_w = BLOCK_SIZE_X * blockIdx.x + threadIdx.x;
    
    int ymax = h - blockIdx.y * BLOCK_SIZE_Y;
    if (ymax > BLOCK_SIZE_Y) ymax = BLOCK_SIZE_Y;        

    for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {
      int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
      s_buf[BLOCK_SIZE_X * y + threadIdx.x] = d_img[offset + w * y + global_w];        
    }
    
  for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {      
      int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
        int idx = BLOCK_SIZE_X * y + threadIdx.x;            
        s_buf2[idx] = idx;
    }
    
    __syncthreads();
    
  for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {

        int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
        int x = threadIdx.x;
        int idx = BLOCK_SIZE_X * y + x;        
        
//        __syncthreads();
              
        // process N
        if (global_w < w && y != 0 && y < ymax && is_conn(s_buf[idx], s_buf[idx - BLOCK_SIZE_X], t)) {
          findAndUnion(s_buf2, idx, idx - BLOCK_SIZE_X);
        }

        __syncthreads();        
        
        // process W
        if (global_w < w && x != 0 && y < ymax && is_conn(s_buf[idx], s_buf[idx -  1], t)) {
          findAndUnion(s_buf2, idx, idx - 1);
        }
        
    }
    
    __syncthreads();
    
    // copy result back
    for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {
      if (global_w >= w) continue;
      int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
        int idx = BLOCK_SIZE_X * y + threadIdx.x;
        
//        int v = s_buf2[idx];
        int v = find(s_buf2, idx);
        
        // convert v to global space
        int v_x = v % BLOCK_SIZE_X;
        int v_y = v / BLOCK_SIZE_X;
        
        d_dst_img[offset + w * y + global_w] = offset + blockIdx.x * BLOCK_SIZE_X + v_y * w + v_x;
    }    
    
}


__global__ void calculate_neighbours8_cuda(int* d_img, int* d_dst_img, int w, int t, int h) {

  extern __shared__ int s_buf[];    
    
    int* s_buf2 = &s_buf[BLOCK_SIZE_Y*BLOCK_SIZE_X];
    
    int offset = BLOCK_SIZE_Y * blockIdx.y * w;    
    int global_w = BLOCK_SIZE_X * blockIdx.x + threadIdx.x;
    
    int ymax = h - blockIdx.y * BLOCK_SIZE_Y;
    if (ymax > BLOCK_SIZE_Y) ymax = BLOCK_SIZE_Y;        

    for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {
      int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
      s_buf[BLOCK_SIZE_X * y + threadIdx.x] = d_img[offset + w * y + global_w];        
    }
    
  for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {      
      int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
        int idx = BLOCK_SIZE_X * y + threadIdx.x;            
        s_buf2[idx] = idx;
    }
    
    __syncthreads();
    
  for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {

        int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
        int x = threadIdx.x;
        
        int idx = BLOCK_SIZE_X * y + x;        
        
//        __syncthreads();
      
        // process NW
        if (global_w < w && x != 0 && y != 0 && y < ymax && is_conn(s_buf[idx], s_buf[idx - BLOCK_SIZE_X - 1], t)) {
          findAndUnion(s_buf2, idx, idx - BLOCK_SIZE_X - 1);
        }
        
        __syncthreads();
        
        // process N
        if (global_w < w && y != 0 && y < ymax && is_conn(s_buf[idx], s_buf[idx - BLOCK_SIZE_X], t)) {
          findAndUnion(s_buf2, idx, idx - BLOCK_SIZE_X);
        }

//        __syncthreads();        
        
        // process NE
        if (global_w < w-1 && y != 0 && (x != BLOCK_SIZE_X - 1) && y < ymax && is_conn(s_buf[idx], s_buf[idx - BLOCK_SIZE_X + 1], t)) {
          findAndUnion(s_buf2, idx, idx - BLOCK_SIZE_X + 1);
        }
        
        __syncthreads();        
        
        // process W
        if (global_w < w && x != 0 && y < ymax && is_conn(s_buf[idx], s_buf[idx -  1], t)) {
          findAndUnion(s_buf2, idx, idx - 1);
        }
        
    }
    
    __syncthreads();
    
    // copy result back
    for(int i=0; i<LOCAL_CCL_Y_IT; ++i) {
      if (global_w >= w) continue;
        
      int y = (threadIdx.y + LOCAL_CCL_Y_STEP*i);
        int idx = BLOCK_SIZE_X * y + threadIdx.x;
        
        int v = find(s_buf2, idx);
        
        // convert v to global space
        int v_x = v % BLOCK_SIZE_X;
        int v_y = v / BLOCK_SIZE_X;
      d_dst_img[offset + w * y + BLOCK_SIZE_X * blockIdx.x + threadIdx.x] = offset + blockIdx.x * BLOCK_SIZE_X + v_y * w + v_x;
    }    
    
}

void debugTime(char* label, Timer& timer) {
    printf("%s: %f\n", label, timer.getTimeElapsed());
}

void debugTime(char* label, int number, Timer& timer) {
    printf("%s %d: %f\n", label, number, timer.getTimeElapsed());
}

int roundUp(int v, int a) {
    if (v%a==0) return v;
    return v + a - v % a;
}

int divRoundUp(int a, int b) {
  int rounding = (a % b == 0) ? 0 : 1;
    return a / b + rounding;
}

void initDevice() {
  
    int count = 0;
    hipGetDeviceCount(&count);
/*  printf("devices: %d\n", count);

    for(int i=0; i<count; ++i) {
      hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("device %d = %s\n", i, prop.name);
    }*/
    hipSetDevice(0);
    checkCUDAError("init");
}


void CCL::cuda_neighbour_process() {

  Timer t_k_neigh, t_k_seams, t_k_final;
    Timer t_cpy_to, t_cpy_from;
  Timer t_cuda;
    Timer t_alloc, t_free;
    
    Timer t;
    t.start();

#ifdef DEVICE_QUERY
    hipDeviceProp_t props;    
    hipGetDeviceProperties(&props, 0);
    printf("totalGlobalMem=%d\nsharedMemPerBlock=%d\nregsPerBlock=%d\n", props.totalGlobalMem, props.sharedMemPerBlock, props.regsPerBlock);
    printf("cap=%d.%d canMapHostMemory=%d deviceOvelap=%d\n", props.major, props.minor, props.canMapHostMemory, props.deviceOverlap);
  checkCUDAError("propquery");
#endif    

  initDevice();

//    const int BLOCK_SIZE = 32;
    
    int *p_img_d_unaligned = 0;
    int *p_img_dst = 0;    
    
    int img_size = m_W * m_H * sizeof(int);
    int img_unaligned_size = (roundUp(m_H, BLOCK_SIZE_Y) * m_W + BLOCK_SIZE_X) * sizeof(int);

  t_alloc.start();
    hipMalloc( (void**)&p_img_d_unaligned, img_unaligned_size );
    hipMalloc( (void**)&p_img_dst, img_unaligned_size );    
    checkCUDAError("malloc0");
    t_alloc.stopAndAddTime();
    
    printf("after alloc: %f\n", t.getTimeElapsed());
    
    // copy BLOCK_SIZE lines to the device
    t_cpy_to.start();
    hipMemcpy( &p_img_d_unaligned[0], &(*m_result)[0], img_size, hipMemcpyHostToDevice);
  checkCUDAError("copy hostToDevice");        
    t_cpy_to.stopAndAddTime();
    
    int yBlocks = divRoundUp(m_H, BLOCK_SIZE_Y);
    int xBlocks = divRoundUp(m_W, BLOCK_SIZE_X);

  t_cuda.start();
    
    printf("before kernels: %f\n", t.getTimeElapsed());

    // start local ccl calculation for blocks
  t_k_neigh.start();    
    dim3 dimGrid(xBlocks, yBlocks);
    dim3 dimBlock(BLOCK_SIZE_X, LOCAL_CCL_THREADS/BLOCK_SIZE_X);
  
    int s_mem = sizeof(int) * BLOCK_SIZE_X * BLOCK_SIZE_Y * 2;

    if (m_diag == false) {
        calculate_neighbours4_cuda<<< dimGrid, dimBlock, s_mem >>>(&p_img_d_unaligned[0], &p_img_dst[0], m_W, m_thresh, m_H);
    } else {    
        calculate_neighbours8_cuda<<< dimGrid, dimBlock, s_mem >>>(&p_img_d_unaligned[0], &p_img_dst[0], m_W, m_thresh, m_H);
    }
  hipDeviceSynchronize();    
    t_k_neigh.stopAndAddTime();
    
    // join the local ccl calculation to global
    t_k_seams.start();
    dim3 dimBlockJoin(max(BLOCK_SIZE_Y, BLOCK_SIZE_X));
    if (m_diag == false) {
        join_seams4_cuda<<< dimGrid, dimBlockJoin >>>(&p_img_d_unaligned[0], &p_img_dst[0], m_W, m_thresh, m_H);
    } else {
        join_seams8_cuda<<< dimGrid, dimBlockJoin >>>(&p_img_d_unaligned[0], &p_img_dst[0], m_W, m_thresh, m_H);
    }
  hipDeviceSynchronize();    
    t_k_seams.stopAndAddTime();

  // final pass
    t_k_final.start();    
    dim3 dimFinalBlock(FINAL_BLOCK_SIZE_X, FINAL_PASS_THREADS/FINAL_BLOCK_SIZE_X);
    dim3 dimFinalGrid(divRoundUp(m_W, FINAL_BLOCK_SIZE_X), divRoundUp(m_H, FINAL_BLOCK_SIZE_Y));
    final_find_cuda<<< dimFinalGrid, dimFinalBlock >>>(&p_img_dst[0], m_W, m_H);    
    hipDeviceSynchronize();
    t_k_final.stopAndAddTime();
    
    printf("after kernels: %f\n", t.getTimeElapsed());
    
    t_cuda.stopAndAddTime();
 
//  t_wait_for_sync.start();
//  hipDeviceSynchronize();
//  t_wait_for_sync.stopAndAddTime();
                
    // copy image from the device
    t_cpy_from.start();
    hipMemcpy( &(*m_result)[0], &p_img_dst[0], img_size, hipMemcpyDeviceToHost);
  checkCUDAError("copy deviceToHost");    
    t_cpy_from.stopAndAddTime();
 
  t_free.start();
    hipFree(p_img_d_unaligned);
    hipFree(p_img_dst);
    t_free.stopAndAddTime();
#ifdef PRINT_TIMINGS    
    printf("time for cuda_neighbour_process(): %f\n", t.getTimeElapsed());
    printf(" thereof alloc:          %f\n", t_alloc.getTimeElapsed());    
    printf(" thereof copy  to:       %f\n", t_cpy_to.getTimeElapsed());    
    printf(" thereof kernel neigh:   %f\n", t_k_neigh.getTimeElapsed());
    printf(" thereof kernel seams:   %f\n", t_k_seams.getTimeElapsed());    
    printf(" thereof kernel final:   %f\n", t_k_final.getTimeElapsed());    
//    printf(" thereof wait for sync:  %f\n", t_wait_for_sync.getTimeElapsed());
    printf(" thereof copy  from:     %f\n", t_cpy_from.getTimeElapsed());
    printf(" thereof free:           %f\n", t_free.getTimeElapsed());
    printf("total gpu time:          %f\n", t_cuda.getTimeElapsed());
#endif

}


int main( int, char** ) {
  CCL ccl;
  QImage image1( "../inputs/cerberus_enhanced.bmp" );
  vector <int> image;
  for (int i=0; i<image1.width(); ++i ) {
    for (int j=0; j<image1.height(); ++j ) {
      image.push_back( image1.pixel(i,j) );
    }
  }
  vector <int> result = ccl.cuda_ccl( image, image1.width() , 4, 42);
  QImage image2;
  for (int i=0; i<image1.width(); ++i ) {
    for (int j=0; j<image1.height(); ++j ) {
      image2.setPixel( i, j, result[ (j*image1.width())+j] );
    }
  }
  image2.save("result.jpg");
}